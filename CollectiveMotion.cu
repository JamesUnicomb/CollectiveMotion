#include "hip/hip_runtime.h"
#include <math.h>
#define FLOATMAX 1e10
#define EPS2 0.000001


__global__ void update(float3 *pos, 
                       float3 *vel, 
                       float3 *pos_, 
                       float3 *vel_, 
                       float3 *noise, 
                       int n, 
                       float timedelta,
                       float ra = 0.8, 
                       float rb = 0.2, 
                       float re = 0.5, 
                       float r0 = 1.0, 
                       float b = 5.0, 
                       float J = 0.001)
{
    float d[nc] = {[0 ... (nc-1)] = FLOATMAX};
    int ni[nc];

    int id = threadIdx.x + blockDim.x*blockIdx.x;

    for (int j = 0; j < nc; j++)
    {
        for (int sub_id = 0; sub_id < n; sub_id++)
        {
            float3 r;

            r.x = pos_[sub_id].x - pos_[id].x;
            r.y = pos_[sub_id].y - pos_[id].y;
            r.z = pos_[sub_id].z - pos_[id].z;

            float dist = sqrtf(r.x * r.x + r.y * r.y + r.z * r.z);

            if ((dist < d[j]) && (id != sub_id) && (dist > d[j-1]))
            {
                d[j] = dist;
                ni[j] = sub_id;
            }
            else if ((dist < d[0]) && (id != sub_id) && (j == 0))
            {
                d[j] = dist;
                ni[j] = sub_id;
            }
        }
    }

    
    float3 cohesion;
    float3 alignment;

    for (int k = 0; k < nc; k++)
    {
        float f;
        float3 r;

        r.x = pos_[ni[k]].x - pos_[id].x;
        r.y = pos_[ni[k]].y - pos_[id].y;
        r.z = pos_[ni[k]].z - pos_[id].z;

        float dist = sqrtf(r.x * r.x + r.y * r.y + r.z * r.z);

        if (dist < rb) 
        {
            f = -1.0 * FLOATMAX;
        }
        else if ((rb <= dist) && (dist < ra)) 
        {
            f = 0.25 * ((dist - re) / (ra - re));
        }
        else if ((ra <= dist) && (dist < r0)) 
        {
            f = 1.0;
        }
        else if ((r0 <= dist))
        {
            f = 0.0;
        }

        cohesion.x += f * r.x;
        cohesion.y += f * r.y;
        cohesion.z += f * r.z;

        alignment.x += vel_[ni[k]].x;
        alignment.y += vel_[ni[k]].y;
        alignment.z += vel_[ni[k]].z;
    }

    vel[id].x = J * (float)(nc) * alignment.x + b * cohesion.x + (float)(nc) * noise[id].x;
    vel[id].y = J * (float)(nc) * alignment.y + b * cohesion.y + (float)(nc) * noise[id].y;
    vel[id].z = J * (float)(nc) * alignment.z + b * cohesion.z + (float)(nc) * noise[id].z;

    float V = sqrtf(vel[id].x * vel[id].x + vel[id].y * vel[id].y + vel[id].z * vel[id].z + EPS2);

    vel[id].x /= V;
    vel[id].y /= V;
    vel[id].z /= V;

    pos[id].x = pos_[id].x + timedelta * vel[id].x;
    pos[id].y = pos_[id].y + timedelta * vel[id].y;
    pos[id].z = pos_[id].z + timedelta * vel[id].z;
}